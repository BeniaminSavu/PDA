#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>

void saveBmp(char* filename, unsigned char *image, int width, int height){
	FILE* f = fopen("test.bmp", "rb");
	unsigned char info[54];
	fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header
	fclose(f);
	
	f = fopen(filename, "wb");
	fwrite(info, sizeof(unsigned char), 54, f);
	unsigned char tmp;

	int size = 3 * width * height;
	unsigned char *data = new unsigned char[size];
	for (int i = 0, k=0; i < size; i+=3, k++){
		data[i] = data[i + 1] = data[i + 2] = image[k];
	}
	fwrite(data, sizeof(unsigned char), size, f);
	fclose(f);
}

unsigned char* readBMP(char* filename, int *width, int *height)
{
	FILE* f = fopen(filename, "rb");
	unsigned char info[54];
	fread(info, sizeof(unsigned char), 54, f); // read the 54-byte header

	// extract image height and width from header
	*width= *(int*)&info[18];
	*height = *(int*)&info[22];

	int size = 3 * (*width) * (*height);
	unsigned char *data = new unsigned char[size]; // allocate 3 bytes per pixel
	fread(data, sizeof(unsigned char), size, f); // read the rest of the data at once
	fclose(f);

	int size2 = *width * (*height);
	unsigned char *redCol = new unsigned char[size2];
	int k = 0;
	for (int i = 2; i < size; i += 3){
		redCol[k] = data[i];
		k++;
	}

	return redCol;
}

__global__ void edgeDetection(unsigned char *data, unsigned char *newData, int *width, int *height)
{
	int pixelMatrix[3][3];
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	int row = i / (*width);
	int col = i % (*width);

	if (row > 0 && row < *height - 1 && col>0 && col < *width - 1){
		pixelMatrix[0][0] = data[(row - 1) * (*width) + col - 1];
		pixelMatrix[0][1] = data[(row - 1) * (*width) + col];
		pixelMatrix[0][2] = data[(row - 1) * (*width) + col + 1];
		pixelMatrix[1][0] = data[row * (*width) + col - 1];
		pixelMatrix[1][2] = data[row * (*width) + col + 1];
		pixelMatrix[2][0] = data[(row + 1) * (*width) + col - 1];
		pixelMatrix[2][1] = data[(row + 1) * (*width) + col];
		pixelMatrix[2][2] = data[(row + 1) * (*width) + col + 1];

		double gy = pixelMatrix[0][0] * (-1) + pixelMatrix[0][1] * (-2) + pixelMatrix[0][2] * (-1) + pixelMatrix[2][0] + pixelMatrix[2][1] * 2 + pixelMatrix[2][2] * 1;
		double gx = pixelMatrix[0][0] + pixelMatrix[0][2] * (-1) + pixelMatrix[1][0] * 2 + pixelMatrix[1][2] * (-2) + pixelMatrix[2][0] + pixelMatrix[2][2] * (-1);
		int gradient = (int)sqrt(gy*gy + gx*gx);

		newData[i] = gradient;
	}else {
		newData[i] = 0;
	}

}

int main()
{	
	int width, height;
	unsigned char *h_data = readBMP("test.bmp", &width, &height);
	
	unsigned char *d_data = 0;
	unsigned char *d_newData = 0;
	int *d_width;
	int *d_height;

	size_t size = height * width;
	unsigned char *h_newData = new unsigned char[size];
	
	hipSetDevice(0);
	hipMalloc(&d_data, size);
	hipMalloc(&d_width, sizeof(int));
	hipMalloc(&d_height, sizeof(int));
	hipMalloc(&d_newData, size * sizeof(unsigned char));
	hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
	hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);

	edgeDetection << <width, height >> >(d_data, d_newData, d_width, d_height);
	hipDeviceSynchronize();
	hipMemcpy(h_newData, d_newData, size, hipMemcpyDeviceToHost);

	

	saveBmp("test2.bmp", h_newData, width, height);
    return 0;
}

