#include "hip/hip_runtime.h"
#include <stddef.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""


#define N 6


__global__ void RoyFloyd(int a[N][N], int k)
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	if (a[i][j] > a[i][k] + a[k][j])
		a[i][j] = a[i][k] + a[k][j];
}

int main()
{
	int h_a[N][N] = {
		{ 0, 2, 5, 999, 999, 999 },
		{ 999, 0, 7, 1, 99 , 8 },
		{ 999, 999, 0, 4, 999, 999 },
		{ 999, 999, 999, 0, 3, 999 },
		{ 999, 999, 2, 999, 0, 3 },
		{ 999, 5, 999, 2, 4, 0 }
	};
	size_t size = N * sizeof(int);
	int *d_a;
	int *d_k;
	hipMalloc(&d_a, size);
	hipMalloc(&d_k, sizeof(int));

	int numBlocks = 1;
	dim3 threadsPerBlock(N, N);
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	for (int k = 0; k < N; k++) {
		hipMemcpy(d_k, &k, sizeof(int), hipMemcpyHostToDevice);
		RoyFloyd <<<numBlocks, threadsPerBlock >>> (d_a, d_k);
	}
	hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_k);
    return 0;
}

